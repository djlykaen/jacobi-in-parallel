#include "hip/hip_runtime.h"
#include "stdafx.h"

#include <hip/hip_runtime.h>
#include <cutil_inline.h>

int carry(int dividend, int divsor)
{
	int quotient = dividend / divsor;
	return quotient * divsor == dividend ? quotient : quotient + 1;
}

//*****************************************************************************
//Cuda jacobi with FLOAT
//*****************************************************************************
//init non-boundarys
__global__ void 
kernelInitInner_f(float* u, float* w, float average, int x, int y)
{
	int id = blockIdx.x * blockDim.x +threadIdx.x;
	if (id < x * y)
	{
		u[id] = average; w[id] = average;
	}
}

//init up & down boundarys
__global__ void kernelInitUpDownBoundary_f(float* u, float* w, 
									       float up, float down, int x, int y)
{
	int id = blockIdx.x * blockDim.x +threadIdx.x;
		if (id < y - 1)
	{
		int upLoca = id;
		u[upLoca] = up;	w[upLoca] = up;
		int downLoca = x * y - y + upLoca;
		u[downLoca] = down;	w[downLoca] = down;
	}
}

//init left & right boundarys
__global__ void kernelInitLeftRightBoundary_f(float* u, float* w, 
										float left, float right, int x, int y)
{
	int id = blockIdx.x * blockDim.x +threadIdx.x;
	if (id < x - 1)
	{
		int leftLoca = y + id * y;
		u[leftLoca] = left;	w[leftLoca] = left;
		int rightLoca = leftLoca - 1;
		u[rightLoca] = right; w[rightLoca] = right;
	}
}

//init with float
void cudaInitMatrix_f(int x, int y, 
				      float left, float up, float right, float down, 
				      float *d_u, float *d_w)
{
	int			size_block = 256;
	dim3		block(size_block);
	dim3		gridLine(carry(y - 1, size_block));
	dim3		gridColumn(carry(x - 1, size_block));
	dim3		gridInner(carry(x * y, size_block));

	float		average = ((x - 2) * (left + right) + (y - 2) * (up + down))
										/ ((x + y - 4) * 2);
	kernelInitInner_f<<<gridInner, block>>>(d_u, d_w, average, x, y);
	kernelInitUpDownBoundary_f<<<gridLine, block>>>(d_u, d_w, up, down, x, y);
	kernelInitLeftRightBoundary_f<<<gridColumn, block>>>(d_u, d_w, left, right, x, y);
}

//kernel of jacobi iteration process with float in iteration mode
__global__ void kernelJacobiIteration_f(int x, int y, float *d_u, float *d_w)
{
	int id = blockIdx.x * blockDim.x +threadIdx.x;
	int t = id / y;
	if ( id < x * y - 2 * y -1 && id != t * y   && id != t * y + y - 1)
	{
		id = id + y;	
		d_w[id] = (d_u[id - y] + d_u[id + y] + d_u[id - 1] 
											+ d_u[id + 1]) / 4.0;
	}	
}

//kernel of getting all epsilon between d_u and d_w
__global__ void kernelGetAllEpsilon_f(int x, int y, 
									  float *d_u, float *d_w, float *d_ep)
{
	int id = blockIdx.x * blockDim.x +threadIdx.x;
	if ( id < x * y - 2 * y -1)
	{
		id = id + y;	
		d_ep[id - y] = d_u[id] - d_w[id];
		if (d_ep[id - y] < 0) d_ep[id - y] = - d_ep[id - y];
	}	
}

//kernel of max of a num group
__global__ void kernelMaxofGroup_f(int count, float *d_ep)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (d_ep[id] < d_ep[id + count]) d_ep[id] = d_ep[id + count];
}

float getEpsilon_f(int x, int y, float *d_u, float *d_w)
{
	float		epsilon;
	float		*d_ep;
    cutilSafeCall(hipMalloc((void**) &d_ep, (x * y - 2 * y) * sizeof(float)));
	int			size_block = 256;
    dim3		block(size_block);
    dim3		grid(carry(x * y - 2 * y , size_block));
	dim3		grid2(carry(x * y / 2 - y, size_block));
	kernelGetAllEpsilon_f<<<grid, block>>>(x, y, d_u, d_w, d_ep);
	//get max of d_ep
	int			count = (int)(x * y / 2 - y);
	while(count > 1)
	{
		kernelMaxofGroup_f<<<grid2, block>>>(count, d_ep);
		count = (count + 1)/ 2;
	}

	cutilSafeCall(hipMemcpy(&epsilon, d_ep, sizeof(float), hipMemcpyDeviceToHost));
	cutilSafeCall(hipFree(d_ep));

	return epsilon;
}

//jacobi iteration process with float in epsilon mode
void cudaJacobiIteration_epsilon_f(int x, int y, float epsilon, 
								   float *d_u, float *d_w, int *iteration)
{
	float		*temp;	
	// setup execution parameters	
	int			size_block = 256;
    dim3		block(size_block);
    dim3		grid(carry(x * y - 2 * y , size_block));
	int			step = 0;
	float		goal_epsilon = epsilon + 1;
    
	//iteration
    while (goal_epsilon > epsilon)
	{
		step = step + 1;
		for (int i = 0; i < 1; i ++)
		{
			//execute the kernel
			kernelJacobiIteration_f<<<grid, block>>>(x, y, d_u, d_w);
			//check if kernel execution generated and error
			cutilCheckMsg("Kernel execution failed");
			temp = d_u; d_u = d_w; d_w = temp;
		}
		//epsilon
		goal_epsilon = getEpsilon_f(x, y, d_u, d_w);		
    }
	*iteration = step;
}

//jacobi iteration process with float in iteration mode
void cudaJacobiIteration_iteration_f(int x, int y, int iteration, 
									 float *d_u, float *d_w, float *epsilon)
{
	float		*temp;
	float		*d_ep;	
    cutilSafeCall(hipMalloc((void**) &d_ep, (x * y - 2 * y) * sizeof(float)));
	// setup execution parameters	
	int			size_block = 256;
    dim3		block(size_block);
    dim3		grid(carry(x * y - 2 * y, size_block));
	dim3		grid2(carry(x * y / 2 - y, size_block));
    
	//iteration
    for (int i = 0; i < iteration; i ++)
    {
        // execute the kernel
        kernelJacobiIteration_f<<<grid, block>>>(x, y, d_u, d_w);
		// check if kernel execution generated and error
		cutilCheckMsg("Kernel execution failed");
		temp = d_u; d_u = d_w; d_w = temp;
    }
	//epsilon
	*epsilon = getEpsilon_f(x, y, d_u, d_w);	
}

//cuda jacobi with float
void cudaJacobi_f(int argc, char** argv, 
				  float left, float up, float right, float down, 
				  int x, int y,
				  float epsilon, 
				  int iteration, 
				  const char *outputfilename)
{
    if( cutCheckCmdLineFlag(argc, (const char**)argv, "device") )
        cutilDeviceInit(argc, argv);
    else
        hipSetDevice(cutGetMaxGflopsDeviceId());

	// create and start timer
    unsigned int	timer = 0;
    cutilCheckError(cutCreateTimer(&timer));
    cutilCheckError(cutStartTimer(timer));

    // allocate host memory for matrices u
    unsigned int	size_u = x * y;
    unsigned int	mem_size_u = sizeof(float) * size_u;

    // allocate device memory
    float*			d_u;
    cutilSafeCall(hipMalloc((void**) &d_u, mem_size_u));
    float*			d_w;
    cutilSafeCall(hipMalloc((void**) &d_w, mem_size_u));

	//init
	cudaInitMatrix_f(x, y, left, up, right, down, d_u, d_w);

	//jacobi iteration process in epsilon mode
	if (epsilon != 0 && iteration == 0)
	{
		outputString(" - Epsilon Mode ", true);
		outputFloat(epsilon, "\tEpsilon = ",true);
		outputString("\tComputing starts...", false);

		cudaJacobiIteration_epsilon_f(x, y, epsilon, d_u, d_w, &iteration);

		outputString("Computing ends.", true);
		outputInt(iteration, "\tIteration count = ", true);
	}
	//jacobi iteration process in iteration mode
	else if (epsilon == 0 && iteration != 0)
	{
		outputString(" - Iteration Mode", true);
		outputInt(iteration, "\tIteration = ",true);
		outputString("\tComputing starts...", false);

		cudaJacobiIteration_iteration_f(x, y, iteration, d_u, d_w, &epsilon);

		outputString("Computing ends.", true);
		outputFloat(epsilon, "\tEpsilon = ", true);
	}
	else
		outputString("\tEpsilon or Iteration is wrong.", true);

    // copy result from device to host	
    float*			u = (float*) malloc(mem_size_u);
    cutilSafeCall(hipMemcpy(u, d_u, mem_size_u,
                              hipMemcpyDeviceToHost) );

    // stop and destroy timer
    cutilCheckError(cutStopTimer(timer));
	outputDouble(cutGetTimerValue(timer),"Processing time (ms): ", true);
    //printf("Processing time: %f (ms) \n", cutGetTimerValue(timer));
    cutilCheckError(cutDeleteTimer(timer)); 

	//output matix
	//outputMatrix_f(u, x, y, "matrix from GPU :", false);
	outputFloatMatrixtoFile(u, x, y, "matrix from GPU-CUDA with Float :", outputfilename);

    // clean up memory
    free(u);
    cutilSafeCall(hipFree(d_u));
    cutilSafeCall(hipFree(d_w));

    hipDeviceReset();
}

//*****************************************************************************
//Cuda jacobi with DOUBLE
//*****************************************************************************
//test double 
__global__ void 
kernelTest_d(double* u)
{
	int id = blockIdx.x * blockDim.x +threadIdx.x;
	u[id] = 0; 
}

//init non-boundarys
__global__ void 
kernelInitInner_d(double* u, double* w, double average, int x, int y)
{
	int id = blockIdx.x * blockDim.x +threadIdx.x;
	if (id < x * y)
	{
		u[id] = average; w[id] = average;
	}
}

//init up & down boundarys
__global__ void kernelInitUpDownBoundary_d(double* u, double* w, 
									       double up, double down, int x, int y)
{
	int id = blockIdx.x * blockDim.x +threadIdx.x;
		if (id < y - 1)
	{
		int upLoca = id;
		u[upLoca] = up;	w[upLoca] = up;
		int downLoca = x * y - y + 1 + upLoca;
		u[downLoca] = down;	w[downLoca] = down;
	}
}

//init left & right boundarys
__global__ void kernelInitLeftRightBoundary_d(double* u, double* w, 
											  double left, double right, int x, int y)
{
	int id = blockIdx.x * blockDim.x +threadIdx.x;
	if (id < x - 1)
	{
		int leftLoca = y + id * y;
		u[leftLoca] = left;	w[leftLoca] = left;
		int rightLoca = leftLoca - 1;
		u[rightLoca] = right; w[rightLoca] = right;
	}
}

//init with double
void cudaInitMatrix_d(int x, int y, 
				      double left, double up, double right, double down, 
				      double *d_u, double *d_w)
{
	int			size_block = 256;
	dim3		block(size_block);
	dim3		gridLine(carry(y - 1, size_block));
	dim3		gridColumn(carry(x - 1, size_block));
	dim3		gridInner(carry(x * y, size_block));

	double		average = ((x - 2) * (left + right) + (y - 2) * (up + down))
										/ ((x + y - 4) * 2);
	//outputDouble(average, "Average = ", true); getchar();
	//dim3		gridTest(1);
	//kernelTest_d<<<gridTest, block>>>(d_u);
	kernelInitInner_d<<<gridInner, block>>>(d_u, d_w, average, x, y);
	kernelInitLeftRightBoundary_d<<<gridColumn, block>>>(d_u, d_w, left, right, x, y);
	kernelInitUpDownBoundary_d<<<gridLine, block>>>(d_u, d_w, up, down, x, y);
}

//kernel of jacobi iteration process with double in iteration mode
__global__ void kernelJacobiIteration_d(int x, int y, double *d_u, double *d_w)
{
	int id = blockIdx.x * blockDim.x +threadIdx.x;
	int t = id / y;
	if ( id < x * y - 2 * y -1 && id != t * y   && id != t * y + y - 1)
	{
		id = id + y;	
		d_w[id] = (d_u[id - y] + d_u[id + y] + d_u[id - 1] 
											+ d_u[id + 1]) / 4.0;
	}	
}

//kernel of getting all epsilon between d_u and d_w
__global__ void kernelGetAllEpsilon_d(int x, int y, 
									  double *d_u, double *d_w, double *d_ep)
{
	int id = blockIdx.x * blockDim.x +threadIdx.x;
	if ( id < x * y - 2 * y )
	{
		id = id + y;	
		d_ep[id - y] = d_u[id] - d_w[id];
		if (d_ep[id - y] < 0) d_ep[id - y] = - d_ep[id - y];
	}	
}

//kernel of max of a num group
__global__ void kernelMaxofGroup_d(int count, double *d_ep)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < count)
		if (d_ep[id] < d_ep[id + count]) d_ep[id] = d_ep[id + count];
}

double getEpsilon_d(int x, int y, double *d_u, double *d_w)
{
	double		epsilon;
	double		*d_ep;
    cutilSafeCall(hipMalloc((void**) &d_ep, (x * y - 2 * y) * sizeof(double)));
	int			size_block = 256;
    dim3		block(size_block);
    dim3		grid(carry(x * y - 2 * y , size_block));
	dim3		grid2(carry(x * y / 2 - y, size_block));
	kernelGetAllEpsilon_d<<<grid, block>>>(x, y, d_u, d_w, d_ep);
	//// allocate host memory for matrices u
 //   unsigned int	size_u = x * y - 2 * y;
 //   unsigned int	mem_size_u = sizeof(double) * size_u;
 //   double*			ep = (double*) malloc(mem_size_u);
 //   cutilSafeCall(hipMemcpy(ep, d_ep, mem_size_u,
 //                             hipMemcpyDeviceToHost));	
	//outputDoubleMatrixtoFile(ep, x - 2, y, "d_ep = ", "d_ep.txt");
	////outputDoubleMatrix(ep, x, y, "****d_ep = ", true);
	//free(ep);
	//get max of d_ep
	int			count = (int)(x * y / 2 - y);
	while(count > 1)
	{
		kernelMaxofGroup_d<<<grid2, block>>>(count, d_ep);
		count = (count + 1)/ 2;
	}

	cutilSafeCall(hipMemcpy(&epsilon, d_ep, sizeof(double), hipMemcpyDeviceToHost));
	cutilSafeCall(hipFree(d_ep));

	return epsilon;
}

//jacobi iteration process with double in epsilon mode
void cudaJacobiIteration_epsilon_d(int x, int y, double epsilon, 
								   double *d_u, double *d_w, int *iteration)
{
	double		*temp;	
	// setup execution parameters	
	int			size_block = 256;
    dim3		block(size_block);
    dim3		grid(carry(x * y - 2 * y , size_block));
	int			step = 0;
	double		goal_epsilon = epsilon + 1;
    
	//iteration
    while (goal_epsilon > epsilon)
	{
		step = step + 1;
		
		//execute the kernel
		kernelJacobiIteration_d<<<grid, block>>>(x, y, d_u, d_w);
		//check if kernel execution generated and error
		cutilCheckMsg("Kernel execution failed");
		temp = d_u; d_u = d_w; d_w = temp;
		//epsilon
		goal_epsilon = getEpsilon_d(x, y, d_u, d_w);
		//if (step % 1000 == 0) 
		//{
		//	goal_epsilon = getEpsilon_d(x, y, d_u, d_w);
		//	outputDouble(goal_epsilon, "goal_epsilon = ", true);
		//}
    }
	*iteration = step;
}

//jacobi iteration process with double in iteration mode
void cudaJacobiIteration_iteration_d(int x, int y, int iteration, 
									 double *d_u, double *d_w, double *epsilon)
{
	double		*temp;
	double		*d_ep;	
    cutilSafeCall(hipMalloc((void**) &d_ep, (x * y - 2 * y) * sizeof(double)));
	// setup execution parameters	
	int			size_block = 256;
    dim3		block(size_block);
    dim3		grid(carry(x * y - 2 * y, size_block));
	dim3		grid2(carry(x * y / 2 - y, size_block));
    
	//iteration
    for (int i = 0; i < iteration; i ++)
    {
        // execute the kernel
        kernelJacobiIteration_d<<<grid, block>>>(x, y, d_u, d_w);
		// check if kernel execution generated and error
		cutilCheckMsg("Kernel execution failed");
		temp = d_u; d_u = d_w; d_w = temp;
		//epsilon
		if (i % 1000 == 0) 
		{
			outputDouble(getEpsilon_d(x, y, d_u, d_w), "goal_epsilon = ", true);
		}
    }
	//epsilon
	*epsilon = getEpsilon_d(x, y, d_u, d_w);	
}

//cuda jacobi with double
void cudaJacobi_d(int argc, char** argv, 
				  double left, double up, double right, double down, 
				  int x, int y,
				  double epsilon, 
				  int iteration, 
				  const char *outputfilename)
{
	char			*s = (char*)malloc(sizeof(char) * 64);
    if (cutCheckCmdLineFlag(argc, (const char**)argv, "device"))
        cutilDeviceInit(argc, argv);
    else
        hipSetDevice(cutGetMaxGflopsDeviceId());

	// create and start timer
    unsigned int	timer = 0;
    cutilCheckError(cutCreateTimer(&timer));
    cutilCheckError(cutStartTimer(timer));

    // allocate host memory for matrices u
    unsigned int	size_u = x * y;
    unsigned int	mem_size_u = sizeof(double) * size_u;

    // allocate device memory
    double*			d_u;
    cutilSafeCall(hipMalloc((void**) &d_u, mem_size_u));
    double*			d_w;
    cutilSafeCall(hipMalloc((void**) &d_w, mem_size_u));

	//init
	cudaInitMatrix_d(x, y, left, up, right, down, d_u, d_w);

	//jacobi iteration process in epsilon mode
	if (epsilon != 0 && iteration == 0)
	{
		outputString(" - Epsilon Mode ", true);
		outputDouble(epsilon, "\tEpsilon = ",true);
		outputString("\tComputing starts...", false);

		cudaJacobiIteration_epsilon_d(x, y, epsilon, d_u, d_w, &iteration);

		outputString("Computing ends.", true);
		outputInt(iteration, "\tIteration count = ", true);

		sprintf(s, "matrix from GPU-CUDA with Double - Epsilon Mode Epsilon=%lf Iteration count=%d :", 
										epsilon, iteration);
	}
	//jacobi iteration process in iteration mode
	else if (epsilon == 0 && iteration != 0)
	{
		outputString(" - Iteration Mode", true);
		outputInt(iteration, "\tIteration = ",true);
		outputString("\tComputing starts...", false);

		cudaJacobiIteration_iteration_d(x, y, iteration, d_u, d_w, &epsilon);

		outputString("Computing ends.", true);
		outputDouble(epsilon, "\tEpsilon = ", true);

		sprintf(s, "matrix from GPU-CUDA with Double - Iteration Mode Iteration=%d Epsilon=%lf :", 
										iteration, epsilon);
	}
	else
		outputString("\tEpsilon or Iteration is wrong.", true);

    // copy result from device to host	
    double*			u = (double*) malloc(mem_size_u);
    cutilSafeCall(hipMemcpy(u, d_u, mem_size_u,
                              hipMemcpyDeviceToHost) );

	//output matix
	//outputDoubleMatrix(u, x, y, "matrix from GPU :", false);
	outputDoubleMatrixtoFile(u, x, y, s, outputfilename);

    // stop and destroy timer
    cutilCheckError(cutStopTimer(timer));
    printf("Processing time: %f (ms) \n", cutGetTimerValue(timer));
    cutilCheckError(cutDeleteTimer(timer));    

    // clean up memory
    free(u);
    cutilSafeCall(hipFree(d_u));
    cutilSafeCall(hipFree(d_w));

    hipDeviceReset();
}