#include "hip/hip_runtime.h"
#include "stdafx.h"

#include <hip/hip_runtime.h>
#include <cutil_inline.h>


//*****************************************************************************
//init matrix non-boundarys
__global__ void kernelInitMatrixInner(const int n, const double value,
									  double *m, double *w)
{
	int id = blockIdx.x * blockDim.x +threadIdx.x;
	if (id < n * n)
	{
		m[id] = value;
		w[id] = value;
	}
}

//*****************************************************************************
//init matrix boundarys
__global__ void kernelInitMatrixBoundary(const int n, const boundary b,
										 double *m, double *w)
{
	int id = blockIdx.x * blockDim.x +threadIdx.x;
	if (id < n - 1)
	{
		int leftId = id * n + n;
		m[leftId] = b.left; w[leftId] = b.left;
		int upId = id;
		m[upId] = b.up;	w[upId] = b.up;		
		int rightId = leftId - 1;
		m[rightId] = b.right; w[rightId] = b.right;
		int downId = n * n - n + 1 + upId;
		m[downId] = b.down;	w[downId] = b.down;
	}
}

//*****************************************************************************
//init matrix
void initMatrix(const int n, const struct boundary b, double *d_m, double *d_w)
{
	int				blockSize = 256;
	dim3			block(blockSize);
	dim3			matrixBoundaryGrid(getQuotient(n - 1, blockSize));
	dim3			matrixInnerGrid(getQuotient(n * n, blockSize));

	kernelInitMatrixInner
		<<<matrixInnerGrid, block>>>(n, b.averageValue, d_m, d_w);
	kernelInitMatrixBoundary
		<<<matrixBoundaryGrid, block>>>(n, b, d_m, d_w);
}

//*****************************************************************************
//kernel of jacobi iteration process with double in iteration mode
__global__ void kernelJacobiIteration(const int n, double *m, double *w)
{
	int id = blockIdx.x * blockDim.x +threadIdx.x;
	if (id < (n - 2) * (n - 2))
	{ 
		int row = id / (n - 2);
		int column = id - row * (n - 2);
		int location = (row + 1) * n + (column + 1);
		w[location] = (m[location - 1] + m[location - n] + m[location + 1] 
											+ m[location + n]) / 4.0;
	}
}

//*****************************************************************************
//kernel of getting epsilon between d_m and d_w
__global__ void kernelGetEpsilon(const int n, 
								 double *m, double *w, 
								 double *ep)
{
	int id = blockIdx.x * blockDim.x +threadIdx.x;
	if (id < (n - 2) * (n - 2))
	{
		int row = id / (n - 2);
		int column = id - row * (n - 2);
		int location = (row + 1) * n + (column + 1);
		ep[id] = fabs(m[location] - w[location]);
	}
}

//kernel of max of a num group
__global__ void kernelGetMax(const int count, double *ep)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < count)
		if (ep[id] < ep[id + count]) ep[id] = ep[id + count];
}

//*****************************************************************************
//get epsilon 
double getEpsilon(const int n, double *d_m, double *d_w)
{
	double			epsilon;
	double			*d_ep;
    cutilSafeCall
		(hipMalloc((void**) &d_ep, (n - 2) * (n - 2) * sizeof(double)));
	// setup execution parameters	
	int				blockSize = 256;
    dim3			block(blockSize);
    dim3			matrixInnerGrid(getQuotient((n - 2) * (n - 2), blockSize));
	dim3			getMaxGrid(getQuotient((n - 2) * (n - 2) / 2, blockSize));
	kernelGetEpsilon<<<matrixInnerGrid, block>>>(n, d_m, d_w, d_ep);
	//check if kernel execution generated and error
	//cutilCheckMsg("Kernel execution failed");
	//get max of d_ep
	int			count = (int)((n - 2) * (n - 2) / 2);
	while(count > 1)
	{
		kernelGetMax<<<getMaxGrid, block>>>(count, d_ep);
		//check if kernel execution generated and error
		//cutilCheckMsg("Kernel execution failed");
		count = (count + 1) / 2;
	}

	double			lastD_ep;
	cutilSafeCall(hipMemcpy(&lastD_ep, d_ep + (n - 2) * (n - 2) - 1, 
									sizeof(double), hipMemcpyDeviceToHost));
	cutilSafeCall
		(hipMemcpy(&epsilon, d_ep, sizeof(double), hipMemcpyDeviceToHost));
	cutilSafeCall(hipFree(d_ep));

	if (epsilon < lastD_ep) epsilon = lastD_ep;

	return epsilon;
}

//*****************************************************************************
void jacobiCUDAIterationEpsilon_1D(const int n, const double epsilon, 
								   long *step, const struct boundary b, 
								   double *d_m, double *d_w,
								   double *initTime, double *iterTime)
{
	double			*temp;
	//timer
	LARGE_INTEGER	nStartCounter, nStopCounter;	
	//init data
	printf("--Data initing(%d, %lf).....", n, epsilon);
	//timer starts
	QueryPerformanceCounter(&nStartCounter);
	//init
	initMatrix(n, b, d_m, d_w);
	//timer ends
	QueryPerformanceCounter(&nStopCounter);
	*initTime = getCostTime(nStartCounter, nStopCounter);
	printf("Done.\n");
	//iteration
	printf("--Computing(%d, %lf).....", n, epsilon);
	//timer starts
	QueryPerformanceCounter(&nStartCounter);
	*step = 0;
	double			epsilonTemp = epsilon + 1;
	// setup execution parameters	
	int				blockSize = 256;
    dim3			block(blockSize);
    dim3			matrixInnerGrid(getQuotient((n - 2) * (n - 2), blockSize));
	//iteration
    while (epsilonTemp > epsilon)
	{
		(*step)++;
		//execute the kernel
		kernelJacobiIteration<<<matrixInnerGrid, block>>>(n, d_m, d_w);
		//check if kernel execution generated and error
		//cutilCheckMsg("Kernel execution failed");
		//epsilon
		if (*step % JUMP == 0)
			epsilonTemp = getEpsilon(n, d_m, d_w);
		temp = d_m; d_m = d_w; d_w = temp;
    }
	//timer ends
	QueryPerformanceCounter(&nStopCounter);
	*iterTime = getCostTime(nStartCounter, nStopCounter);
	printf("Done.\n");
	return;
}

//********************************************************************************
void jacobiCUDAIterationStep_1D(const int n, double *epsilon, 
								const long step, const struct boundary b,
								double *d_m, double *d_w,
								double *initTime, double *iterTime)
{
////*****************************************************************************	
//    unsigned int	matrixSize = n * n;
//    unsigned int	matrixMemSize = sizeof(double) * matrixSize;
//	double			*m = (double *)malloc(sizeof(double) * n * n);
////*****************************************************************************

	double			*temp;
	//timer
	LARGE_INTEGER	nStartCounter, nStopCounter;	
	//init data
	printf("--Data initing(%d, %lf).....", n, epsilon);
	//timer starts
	QueryPerformanceCounter(&nStartCounter);
	//init
	initMatrix(n, b, d_m, d_w);
////*****************************************************************************
//	cutilSafeCall(hipMemcpy(m, d_m, matrixMemSize, hipMemcpyDeviceToHost));
//	printf("\nm\n");
//	for(int i = 0; i < 10; i++)
//	{
//		for(int j = 0; j < 6; j++)
//			printf("%10.2lf  ", m[i * n + j]);
//		printf("\n");
//	}
//	cutilSafeCall(hipMemcpy(m, d_w, matrixMemSize, hipMemcpyDeviceToHost));
//	printf("\nw\n");
//	for(int i = 0; i < 10; i++)
//	{
//		for(int j = 0; j < 6; j++)
//			printf("%10.2lf  ", m[i * n + j]);
//		printf("\n");
//	}
//	getchar();
////*****************************************************************************

	//timer ends
	QueryPerformanceCounter(&nStopCounter);
	*initTime = getCostTime(nStartCounter, nStopCounter);
	printf("Done.\n");
	//iteration
	printf("--Computing(%d, %lf).....", n, epsilon);
	//timer starts
	QueryPerformanceCounter(&nStartCounter);
	// setup execution parameters	
	int				blockSize = 256;
    dim3			block(blockSize);
    dim3			matrixInnerGrid(getQuotient((n - 2) * (n - 2), blockSize));


	for(int i = 0; i < step; i++)
	{
		//execute the kernel
		kernelJacobiIteration<<<matrixInnerGrid, block>>>(n, d_m, d_w);
////*****************************************************************************
//	cutilSafeCall(hipMemcpy(m, d_m, matrixMemSize, hipMemcpyDeviceToHost));
//	printf("\nm\n");
//	for(int i = 0; i < 10; i++)
//	{
//		for(int j = 0; j < 6; j++)
//			printf("%10.2lf  ", m[i * n + j]);
//		printf("\n");
//	}
//	cutilSafeCall(hipMemcpy(m, d_w, matrixMemSize, hipMemcpyDeviceToHost));
//	printf("\nw\n");
//	for(int i = 0; i < 10; i++)
//	{
//		for(int j = 0; j < 6; j++)
//			printf("%10.2lf  ", m[i * n + j]);
//		printf("\n");
//	}
//	getchar();
////*****************************************************************************
		temp = d_m; d_m = d_w; d_w = temp;		
	}
	*epsilon = getEpsilon(n, d_m, d_w);
	//timer ends
	QueryPerformanceCounter(&nStopCounter);
	*iterTime = getCostTime(nStartCounter, nStopCounter);
	printf("Done.\n");
	return;
}

//********************************************************************************
void jacobiCUDA_1D(int argc, char** argv,
				   int n, double epsilon, 
				   long step, struct boundary b, char *outFile)
{
	printf("Jacobi CUDA 1D -\n");
	printf("--n=%d, e=%lf, step=%ld\n--LURD: %lf, %lf, %lf, %lf\n",
		n, epsilon, step, b.left, b.up, b.right, b.down);

	//init cuda device
    if (cutCheckCmdLineFlag(argc, (const char**)argv, "device"))
        cutilDeviceInit(argc, argv);
    else
        hipSetDevice(cutGetMaxGflopsDeviceId());
	
    //allocate devce memory for matrices u
    unsigned int	matrixSize = n * n;
    unsigned int	matrixMemSize = sizeof(double) * matrixSize;

    double			*d_m;
	hipMalloc((void**) &d_m, matrixMemSize);
    cutilSafeCall(hipMalloc((void**) &d_m, matrixMemSize));
    double			*d_w;
    cutilSafeCall(hipMalloc((void**) &d_w, matrixMemSize));

	//timer
	LARGE_INTEGER	nStartCounter, nStopCounter;
	double			nTime1, nTime2, nTime3;

	//jacobi serial 1D solution
	if (epsilon != 0)
	{
		printf("--Epsilon mode\n");
		jacobiCUDAIterationEpsilon_1D(n, epsilon, &step, 
										b, d_m, d_w, &nTime1, &nTime2);
		printf("--Step = %ld\n", step);
	}
	else 
	{
		printf("--Step mode\n");
		jacobiCUDAIterationStep_1D(n, &epsilon, step,
										b, d_m, d_w, &nTime1, &nTime2);
		printf("--Epsilon = %lf\n", epsilon);	
	}
	printf("--Result outputing...");

	char			*outDir = getOutDir(n, epsilon, b, step, outFile);
	//timer starts
	QueryPerformanceCounter(&nStartCounter);
	double			*m = (double *)malloc(sizeof(double) * n * n);
	cutilSafeCall(hipMemcpy(m, d_m, matrixMemSize, hipMemcpyDeviceToHost));
	//output result
	outMatrix1DtoF(m, n, outDir);
	//timer2 ends
	QueryPerformanceCounter(&nStopCounter);
	//get time
	nTime3 = getCostTime(nStartCounter, nStopCounter);
	printf("Done.\n");

	printf("--(Time/s)Init=%lf, Computing=%lf, Data-saving=%lf, Total=%lf\n", 
		nTime1, nTime2, nTime3, nTime1 + nTime2 + nTime3);

	outLog(n, epsilon, step, b, nTime1, nTime2, nTime3, outFile, outDir);

	return;
}