#include "hip/hip_runtime.h"
#include "stdafx.h"

#include <hip/hip_runtime.h>
#include <cutil_inline.h>


//*****************************************************************************
//init matrix non-boundarys
__global__ void kernelInitMatrixInner(const int n, const double value,
									  double *m, double *w)
{
	int id = blockIdx.x * blockDim.x +threadIdx.x;
	if (id < n * n)
	{
		m[id] = value;
		w[id] = value;
	}
}

//*****************************************************************************
//init matrix boundarys
__global__ void kernelInitMatrixBoundary(const int n, const boundary b,
										 double *m, double *w)
{
	int id = blockIdx.x * blockDim.x +threadIdx.x;
	if (id < n - 1)
	{
		int leftId = id * n + n;
		m[leftId] = b.left; w[leftId] = b.left;
		int upId = id;
		m[upId] = b.up;	w[upId] = b.up;		
		int rightId = leftId - 1;
		m[rightId] = b.right; w[rightId] = b.right;
		int downId = n * n - n + 1 + upId;
		m[downId] = b.down;	w[downId] = b.down;
	}
}

//*****************************************************************************
//init matrix
void initMatrix(const int n, const struct boundary b, double *d_m, double *d_w)
{
	int				blockSize = 256;
	dim3			block(blockSize);
	dim3			matrixBoundaryGrid(getQuotient(n - 1, blockSize));
	dim3			matrixInnerGrid(getQuotient(n * n, blockSize));

	kernelInitMatrixInner
		<<<matrixInnerGrid, block>>>(n, b.averageValue, d_m, d_w);
	kernelInitMatrixBoundary
		<<<matrixBoundaryGrid, block>>>(n, b, d_m, d_w);
}

//*****************************************************************************
//kernel of jacobi iteration process with double in iteration mode
__global__ void kernelJacobiIteration(const int n, double *m, double *w)
{
	int id = blockIdx.x * blockDim.x +threadIdx.x;
	int row = id / (n - 1);
	int column = id - row * (n - 1);
	int location = (row + 1) * (column + 1);
	w[location] = (m[location - 1] + m[location - n] + m[location + 1] 
										+ m[location + n]) / 4.0;
}

//*****************************************************************************
//kernel of getting epsilon between d_m and d_w
__global__ void kernelGetEpsilon(const int n, 
								 double *m, double *w, 
								 double *ep)
{
	int id = blockIdx.x * blockDim.x +threadIdx.x;
	int row = id / (n - 1);
	int column = id - row * (n - 1);
	int location = (row + 1) * (column + 1);
	ep[id] = fabs(m[location] - w[location]);	
}

//kernel of max of a num group
__global__ void kernelGetMax(const int count, double *ep)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < count)
		if (ep[id] < ep[id + count]) ep[id] = ep[id + count];
}

//*****************************************************************************
//get epsilon 
double getEpsilon(const int n, double *d_m, double *d_w)
{
	double			epsilon;
	double			*d_ep;
    //cutilSafeCall
	(hipMalloc((void**) &d_ep, (n - 1) * (n - 1) * sizeof(double)));
	// setup execution parameters	
	int				blockSize = 256;
    dim3			block(blockSize);
    dim3			matrixInnerGrid(getQuotient((n - 1) * (n - 1), blockSize));
	dim3			getMaxGrid(getQuotient((n - 1) * (n - 1) / 2, blockSize));
	kernelGetEpsilon<<<matrixInnerGrid, block>>>(n, d_m, d_w, d_ep);
	//check if kernel execution generated and error
	//cutilCheckMsg("Kernel execution failed");
	//get max of d_ep
	int			count = (int)((n - 1) * (n - 1) / 2);
	while(count > 1)
	{
		kernelGetMax<<<getMaxGrid, block>>>(count, d_ep);
		//check if kernel execution generated and error
		//cutilCheckMsg("Kernel execution failed");
		count = (count + 1) / 2;
	}

	double			lastD_ep;
	//cutilSafeCall
		(hipMemcpy(&lastD_ep, d_ep + (n - 1) * (n - 1) - 1, 
									sizeof(double), hipMemcpyDeviceToHost));
	//cutilSafeCall
		(hipMemcpy(&epsilon, d_ep, sizeof(double), hipMemcpyDeviceToHost));
	//cutilSafeCall
		(hipFree(d_ep));

	if (epsilon < lastD_ep) epsilon = lastD_ep;

	return epsilon;
}

//*****************************************************************************
void jacobiCUDAIterationEpsilon_1D(const int n, const double epsilon, 
								   long *step, const struct boundary b, 
								   double *d_m, double *d_w,
								   double *initTime, double *iterTime)
{
	double			*temp;
	//timer
	LARGE_INTEGER	nStartCounter, nStopCounter;	
	//init data
	printf("--Data initing(%d, %lf).....", n, epsilon);
	//timer starts
	QueryPerformanceCounter(&nStartCounter);
	//init
	initMatrix(n, b, d_m, d_w);

//*****************************************************************************
	double			*m = (double *)malloc(sizeof(double) * n * n);
	//cutilSafeCall
	(hipMemcpy(m, d_m, matrixMemSize, hipMemcpyDeviceToHost));
	printf("\n");
	for(int i = 0; i < 10; i++)
		for(int j = 0; j < 10; j++)
			printf("%.2lf  ", m[i * n + j]);
	
//*****************************************************************************

	//timer ends
	QueryPerformanceCounter(&nStopCounter);
	*initTime = getCostTime(nStartCounter, nStopCounter);
	printf("Done.\n");
	//iteration
	printf("--Computing(%d, %lf).....", n, epsilon);
	//timer starts
	QueryPerformanceCounter(&nStartCounter);
	*step = 0;
	double			epsilonTemp = epsilon + 1;
	// setup execution parameters	
	int				blockSize = 256;
    dim3			block(blockSize);
    dim3			matrixInnerGrid(getQuotient((n - 1) * (n - 1), blockSize));
	//iteration
    while (epsilonTemp > epsilon)
	{
		(*step)++;
		printf("%d\n", *step);
		//execute the kernel
		kernelJacobiIteration<<<matrixInnerGrid, block>>>(n, d_m, d_w);
		//check if kernel execution generated and error
		//cutilCheckMsg("Kernel execution failed");
		//epsilon
		if (*step % JUMP == 0)
			epsilonTemp = getEpsilon(n, d_m, d_w);
		temp = d_m; d_m = d_w; d_w = temp;
    }
	//timer ends
	QueryPerformanceCounter(&nStopCounter);
	*iterTime = getCostTime(nStartCounter, nStopCounter);
	printf("Done.\n");
	return;
}

//********************************************************************************
void jacobiCUDAIterationStep_1D(const int n, double *epsilon, 
								const long step, const struct boundary b,
								double *d_m, double *d_w,
								double *initTime, double *iterTime)
{
	double			*temp;
	//timer
	LARGE_INTEGER	nStartCounter, nStopCounter;	
	//init data
	printf("--Data initing(%d, %lf).....", n, epsilon);
	//timer starts
	QueryPerformanceCounter(&nStartCounter);
	//init
	initMatrix(n, b, d_m, d_w);
	//timer ends
	QueryPerformanceCounter(&nStopCounter);
	*initTime = getCostTime(nStartCounter, nStopCounter);
	printf("Done.\n");
	//iteration
	printf("--Computing(%d, %lf).....", n, epsilon);
	//timer starts
	QueryPerformanceCounter(&nStartCounter);
	// setup execution parameters	
	int				blockSize = 256;
    dim3			block(blockSize);
    dim3			matrixInnerGrid(getQuotient((n - 1) * (n - 1), blockSize));
	for(int i = 0; i < step; i++)
	{
		//execute the kernel
		kernelJacobiIteration<<<matrixInnerGrid, block>>>(n, d_m, d_w);
		temp = d_m; d_m = d_w; d_w = temp;		
	}
	*epsilon = getEpsilon(n, d_m, d_w);
	//timer ends
	QueryPerformanceCounter(&nStopCounter);
	*iterTime = getCostTime(nStartCounter, nStopCounter);
	printf("Done.\n");
	return;
}

//********************************************************************************
void jacobiCUDA_1D(int argc, char** argv,
				   int n, double epsilon, 
				   long step, struct boundary b, char *outFile)
{
	printf("Jacobi CUDA 1D -\n");
	printf("--n=%d, e=%lf, step=%ld\n--LURD: %lf, %lf, %lf, %lf\n",
		n, epsilon, step, b.left, b.up, b.right, b.down);

	//init cuda device
    if (cutCheckCmdLineFlag(argc, (const char**)argv, "device"))
        cutilDeviceInit(argc, argv);
    else
        hipSetDevice(cutGetMaxGflopsDeviceId());
	
    //allocate devce memory for matrices u
    unsigned int	matrixSize = n * n;
    unsigned int	matrixMemSize = sizeof(double) * matrixSize;

    double			*d_m;
	hipMalloc((void**) &d_m, matrixMemSize);
    //cutilSafeCall(hipMalloc((void**) &d_m, matrixMemSize));
    double			*d_w;
    //cutilSafeCall
		(hipMalloc((void**) &d_w, matrixMemSize));

	//timer
	LARGE_INTEGER	nStartCounter, nStopCounter;
	double			nTime1, nTime2, nTime3;

	//jacobi serial 1D solution
	if (epsilon != 0)
	{
		printf("--Epsilon mode\n");
		jacobiCUDAIterationEpsilon_1D(n, epsilon, &step, 
										b, d_m, d_w, &nTime1, &nTime2);
		printf("--Step = %ld\n", step);
	}
	else 
	{
		printf("--Step mode\n");
		jacobiCUDAIterationStep_1D(n, &epsilon, step,
										b, d_m, d_w, &nTime1, &nTime2);
		printf("--Epsilon = %lf\n", epsilon);	
	}
	printf("--Result outputing...");

	char			*outDir = getOutDir(n, epsilon, b, step, outFile);
	//timer starts
	QueryPerformanceCounter(&nStartCounter);
	double			*m = (double *)malloc(sizeof(double) * n * n);
	//cutilSafeCall
		(hipMemcpy(m, d_m, matrixMemSize, hipMemcpyDeviceToHost));
	//output result
	outMatrix1DtoF(m, n, outDir);
	//timer2 ends
	QueryPerformanceCounter(&nStopCounter);
	//get time
	nTime3 = getCostTime(nStartCounter, nStopCounter);
	printf("Done.\n");

	printf("--(Time/s)Init=%lf, Computing=%lf, Data-saving=%lf, Total=%lf\n", 
		nTime1, nTime2, nTime3, nTime1 + nTime2 + nTime3);

	outLog(n, epsilon, step, b, nTime1, nTime2, nTime3, outFile, outDir);

	return;
}